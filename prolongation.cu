#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"
#include <omp.h>

// Input the coarse gird matrix and calculate the corresponding fine grid matrix by bilinear operator

	// arguments: (1)coarse matrix, (2)matrix size of fine matrix, (3)fine matrix

__global__
void prolongation_gpu( double (*matrix_c), int n_c, double (*matrix_f) ){
	int n_f = 2*n_c-1;
	const int i_f = blockIdx.x;
       	const int j_f = threadIdx.x;
	if( i_f<n_f && j_f<n_f ){	
		matrix_f[i_f*n_f+j_f] = 1.0;	
		/*if( i_f%2==0 && j_f%2==0 ){
			int i_c = i_f/2;
	                int j_c = j_f/2;
        	        matrix_f[i_f*n_f+j_f] = matrix_c[i_c*n_c+j_c];
		}
		__syncthreads();
		if( i_f%2==0 && j_f%2==1 ){
		matrix_f[i_f*n_f+j_f] = ( matrix_f[i_f*n_f+(j_f+1)] + matrix_f[i_f*n_f+(j_f-1)] )/2;
		}
		__syncthreads();
		if( i_f%2==1 ){
		matrix_f[i_f*n_f+j_f] = ( matrix_f[(i_f+1)*n_f+j_f] + matrix_f[(i_f-1)*n_f+j_f] )/2;
		}*/
	}
}


void prolongation( double *matrix_c, int n_c, double *matrix_f) {
#	ifdef OPENMP
	double t;
	t = omp_get_wtime();
#	endif	

	int n_f = 2*n_c-1;
	int i_c, j_c, i_f, j_f;
#	ifdef GPU
	double (*d_matrix_f),(*d_matrix_c);
        hipMalloc( &d_matrix_f, n_f*n_f*sizeof(double));
        hipMalloc( &d_matrix_c, n_c*n_c*sizeof(double));
        hipMemcpy( d_matrix_c, matrix_c, n_c*n_c*sizeof(double), hipMemcpyHostToDevice );
        prolongation_gpu  <<< GRID_SIZE, BLOCK_SIZE >>> ( d_matrix_c, n_c, d_matrix_f );
 	hipMemcpy( matrix_f, d_matrix_f, n_f*n_f*sizeof(double), hipMemcpyDeviceToHost );
        hipFree(d_matrix_f);
        hipFree(d_matrix_c);
        printf("Using gpu prolongate.\n");
	print(matrix_f,n_f);
#	endif



/*
//	Copy the points with factor 1 to the fine matrix
#	ifdef OPENMP
#	pragma omp parallel for collapse(2) private( i_c, j_c ) 
#endif
	for( i_f=0; i_f<n_f; i_f+=2 ) 
	for( j_f=0; j_f<n_f; j_f+=2 ) {
		i_c = i_f/2;
		j_c = j_f/2;
		matrix_f[ind(i_f, j_f, n_f)] = matrix_c[ind(i_c, j_c, n_c)];
	}
//	Compute the rest points
//	Compute even row
#ifdef OPENMP
#	pragma omp parallel for collapse( 2 )
#endif
	for( i_f=0; i_f<n_f; i_f+=2 )
	for( j_f=1; j_f<n_f-1; j_f+=2 ) {
		matrix_f[ind(i_f, j_f, n_f)] = ( matrix_f[ind(i_f, j_f+1, n_f)] + matrix_f[ind(i_f, j_f-1, n_f)] )/2;
	}

//	Compute odd row
#ifdef OPENMP
#	pragma omp parallel for collapse( 2 )
#endif
	for( i_f=1; i_f<n_f-1; i_f+=2 )
	for( j_f=0; j_f<n_f; j_f++ ) {
		matrix_f[ind(i_f, j_f, n_f)] = ( matrix_f[ind(i_f+1, j_f, n_f)] + matrix_f[ind(i_f-1, j_f, n_f)] )/2;
	}
*/	
#	ifdef DEBUG
	t = omp_get_wtime()-t;
	printf("[N_c = %4d -> N_f = %4d] Finish prolongation.(Duration = %.3f sec)\n", n_c, n_f, t);
#	endif

}
