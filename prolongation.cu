#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"
#include <omp.h>

// Input the coarse gird matrix and calculate the corresponding fine grid matrix by bilinear operator

	// arguments: (1)coarse matrix, (2)matrix size of fine matrix, (3)fine matrix

__global__
void prolongation_gpu( double (*matrix_c), int n_c, double (*matrix_f) ){
	int n_f = 2*n_c-1;

	int job = n_f/BLOCK_SIZE+1;

	for( int a=0;a<job;a++ )
	for( int b=0;b<job;b++ ){
	int i_f = blockIdx.x*job+a;
       	int j_f = threadIdx.x*job+b;
	int i_c = i_f/2;
	int j_c = j_f/2;
	if( i_f<n_f && j_f<n_f ){
		if( i_f%2==0 && j_f%2==0 ){
			matrix_f[i_f*n_f+j_f] = matrix_c[i_c*n_c+j_c];
		}else if(i_f%2==0 && j_f%2==1 ){
			j_c = (j_f+1)/2;
			matrix_f[i_f*n_f+j_f] = (matrix_c[i_c*n_c+(j_c-1)]+matrix_c[i_c*n_c+(j_c)])/2;
		}else if(i_f%2==1 && j_f%2==0 ){
			i_c = (i_f+1)/2;
			matrix_f[i_f*n_f+j_f] = (matrix_c[(i_c)*n_c+j_c]+matrix_c[(i_c-1)*n_c+j_c])/2;
		}else if(i_f%2==1 && j_f%2==1 ){
			i_c = (i_f+1)/2;
			j_c = (j_f+1)/2;
			matrix_f[i_f*n_f+j_f] = (matrix_c[(i_c)*n_c+(j_c)]+matrix_c[(i_c-1)*n_c+(j_c-1)]
					+matrix_c[(i_c-1)*n_c+(j_c)]+matrix_c[(i_c)*n_c+(j_c-1)])/4;
		}
	}
	}

}

void prolongation( double *matrix_c, int n_c, double *matrix_f) {
#	ifdef OPENMP
	double t;
	t = omp_get_wtime();
#	endif	

	int n_f = 2*n_c-1;
#	ifdef GPU
	double (*d_matrix_f),(*d_matrix_c);
        hipMalloc( &d_matrix_f, n_f*n_f*sizeof(double));
        hipMalloc( &d_matrix_c, n_c*n_c*sizeof(double));
        hipMemcpy( d_matrix_c, matrix_c, n_c*n_c*sizeof(double), hipMemcpyHostToDevice );
        prolongation_gpu  <<< BLOCK_SIZE, GRID_SIZE >>> ( d_matrix_c, n_c, d_matrix_f );
 	hipMemcpy( matrix_f, d_matrix_f, n_f*n_f*sizeof(double), hipMemcpyDeviceToHost );
        hipFree(d_matrix_f);
        hipFree(d_matrix_c);
#	else
//	Copy the points with factor 1 to the fine matrix
	int i_c, j_c, i_f, j_f;
#	ifdef OPENMP
#	pragma omp parallel for collapse(2) private( i_c, j_c ) 
#	endif//ifdef OPENMP
	for( i_f=0; i_f<n_f; i_f+=2 ) 
	for( j_f=0; j_f<n_f; j_f+=2 ) {
		i_c = i_f/2;
		j_c = j_f/2;
		matrix_f[ind(i_f, j_f, n_f)] = matrix_c[ind(i_c, j_c, n_c)];
	}
//	Compute the rest points
//	Compute even row
#ifdef OPENMP
#	pragma omp parallel for collapse( 2 )
#endif
	for( i_f=0; i_f<n_f; i_f+=2 )
	for( j_f=1; j_f<n_f-1; j_f+=2 ) {
		matrix_f[ind(i_f, j_f, n_f)] = ( matrix_f[ind(i_f, j_f+1, n_f)] + matrix_f[ind(i_f, j_f-1, n_f)] )/2;
	}

//	Compute odd row
#ifdef OPENMP
#	pragma omp parallel for collapse( 2 )
#endif
	for( i_f=1; i_f<n_f-1; i_f+=2 )
	for( j_f=0; j_f<n_f; j_f++ ) {
		matrix_f[ind(i_f, j_f, n_f)] = ( matrix_f[ind(i_f+1, j_f, n_f)] + matrix_f[ind(i_f-1, j_f, n_f)] )/2;
	}
#endif//#ifdef GPU #else
#	ifdef DEBUG
	t = omp_get_wtime()-t;
	printf("[N_c = %4d -> N_f = %4d] Finish prolongation.(Duration = %.3f sec)\n", n_c, n_f, t);
#	endif

}
