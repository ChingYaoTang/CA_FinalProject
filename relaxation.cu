#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"
#include "relative_error.h"
#include <omp.h>

extern const float L;
extern const bool sor_method; 


// arguments: (1)phi matirx, (2)rho matrix, (3)size of the matrix,(4)convergence criterion, 
// 	      (5)updating method: 1="normal", 0="even odd", (6)omega for SOR (should be 1 for smoothing => GS), 
// 	      (7)which equation are we dealing with: 0 for Poisson eq., 1 for residual eq. 

__global__
void relaxation_gpu_odd( double (*phi_guess), double (*rho), int n, double omega, bool w, double h, double *error ){
	int job = n/BLOCK_SIZE+1;

	for( int a=0;a<job;a++ )
	for( int b=0;b<job;b++ ){
	const int i = blockIdx.x*job+a+1;
	const int j = threadIdx.x*job+b+1;
		if( i<n-1 && j<n-1  ){
//	Compute odd cells
		if( (i%2+j%2)%2==0 ){
		 double r = omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
				 		 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4\\
						 -rho[i*n+j]*pow(h,2)*pow(-1,w));
		 error[i*n+j]     = fabs(r/phi_guess[i*n+j]);
		 phi_guess[i*n+j] += r;
		}
		}
	}
}
__global__
void relaxation_gpu_even( double (*phi_guess), double (*rho), int n, double omega, bool w, double h, double *error){
        int job = n/BLOCK_SIZE+1;

        for( int a=0;a<job;a++ )
        for( int b=0;b<job;b++ ){
        const int i = blockIdx.x*job+a+1;
        const int j = threadIdx.x*job+b+1;
                if( i<n-1 && j<n-1 ){
//      Compute odd cells
                if( (i%2+j%2)%2==1 ){
                 double r = omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
                                                 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4\\
                                                 -rho[i*n+j]*pow(h,2)*pow(-1,w));
                 error[i*n+j]     = fabs(r/phi_guess[i*n+j]);
                 phi_guess[i*n+j] += r;
                }
                }
        }
}

__global__
void compute_error( double (*error), int n, double (*result)){
	/*int i = threadIdx.x;
	for( int j=1;j<n;j++ ) error[i*n]+=error[i*n+j];
	__syncthreads();
	if( threadIdx.x == 0 ){
		*result = 0.0;
	for( int k=0;k<n;k++ ){
		*result += error[k*n]/pow(n,2);
	}}
	//error[0] = *result;*/
	*result = 0;
	for( int i=0;i<n;i++ )
	for( int j=0;j<n;j++ ){
		*result = error[i*n+j]/pow(n,2);
	}
}

__global__
void zero( int n, double *error ) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	error[i*n+j] = 0.0;
}




void relaxation( double *phi_guess, double *rho, int n, double *conv_criterion, float omega, bool w ) {
#ifdef DEBUG
	double tr;
	tr	= omp_get_wtime();
#endif
//	Determine the physical grid size
	double h = L/(n-1);

//	Two end criteria for relaxation
	double *itera = (double *)malloc( sizeof(double) );
	*itera = 0;
	double *error = (double *)malloc( sizeof(double) );
	*error = 1;
	double *error_tot;
//	Store the primitive input to make the comparison with the up-to-date result
	double *phi_old = (double *)malloc( n*n*sizeof(double) );

//	Set the end criterion
	double *condition1;
	double *condition2;
	if( *conv_criterion<1.0 ) {
		condition1 = error;
		condition2 = conv_criterion;
	} else {
		condition1 = conv_criterion;
		condition2 = itera;
	}

//	Relaxation
	if( sor_method==1 ) {
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//		       	copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
			for( int i=1; i<(n-1); i++ )
			for( int j=1; j<(n-1); j++ ) {
				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
			    				             + phi_guess[ind(i-1, j, n)]
							             + phi_guess[ind(i, j+1, n)]
							             + phi_guess[ind(i, j-1, n)]
						        	     - phi_guess[ind(i, j, n)]*4
						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
				*error += fabs( ( phi_guess[ind(i, j, n)] - phi_old[ind(i, j, n)] ) / phi_old[ind(i, j, n)] );
			}
		}
	} else if( sor_method==0 ) {
                error_tot = (double *)malloc(n*n*sizeof(double));
#ifdef PARALLEL_GPU
		double (*d_phi), (*d_rho), (*d_error), (*d_result);// (*d_phi_old);
		hipMalloc( &d_phi, n*n*sizeof(double));
                //hipMalloc( &d_phi_old, n*n*sizeof(double));
                hipMalloc( &d_rho, n*n*sizeof(double));
                hipMalloc( &d_error, n*n*sizeof(double));
		hipMalloc( &d_result, sizeof(double));
                //hipMemcpy( d_phi_old, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );
                hipMemcpy( d_phi, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );
                hipMemcpy( d_rho, rho, n*n*sizeof(double), hipMemcpyHostToDevice );

#endif
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//	       		copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
#ifdef PARALLEL_GPU
			relaxation_gpu_odd <<< BLOCK_SIZE,GRID_SIZE >>> ( d_phi, d_rho, n, omega, w, h, d_error);
			relaxation_gpu_even <<< BLOCK_SIZE,GRID_SIZE >>> ( d_phi, d_rho, n, omega, w, h, d_error);
		//	hipMemcpy( error_tot, d_error, n*n*sizeof(double), hipMemcpyDeviceToHost );
			compute_error	<<<1,1>>> ( d_error, n, d_result);
			
		//	hipMemcpy( error_tot, d_error, n*n*sizeof(double), hipMemcpyDeviceToHost );
			hipMemcpy( error, d_result, sizeof(double), hipMemcpyDeviceToHost );
		//	hipMemcpy( phi_guess, d_phi, n*n*sizeof(double), hipMemcpyDeviceToHost );
#endif

#ifdef WO_OMP
		//	printf("Not Using GPU.\n");
//			update odd part
			for( int i=1; i<(n-1); i++ )
 			for( int j=( i%2 + (i+1)%2*2 ); j<(n-1); j+=2 ) {
 				double r = omega/4 * ( phi_guess[ind(i+1, j, n)]
 				    			             + phi_guess[ind(i-1, j, n)]
 							             + phi_guess[ind(i, j+1, n)]
 							             + phi_guess[ind(i, j-1, n)]
 							             - phi_guess[ind(i, j, n)]*4
 						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
				error_tot[i*n+j] = fabs(r/phi_guess[i*n+j]);
				phi_guess[i*n+j] += r;
 			}
//			update even part
 			for( int i=1; i<(n-1); i++ )
 			for( int j=( (i+1)%2 + i%2*2 ); j<(n-1); j+=2 ) {
 				double r = omega/4 * ( phi_guess[ind(i+1, j, n)]
                                                                     + phi_guess[ind(i-1, j, n)]
                                                                     + phi_guess[ind(i, j+1, n)]
                                                                     + phi_guess[ind(i, j-1, n)]
                                                                     - phi_guess[ind(i, j, n)]*4
                                                                     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
                                error_tot[i*n+j] = fabs(r/phi_guess[i*n+j]);
                                phi_guess[i*n+j] += r;
			}
//#endif
			//relative_error( phi_guess, phi_old, n, error );
			for( int i=1;i<(n-1);i++ )
			for( int j=1;j<(n-1);j++ ){
                                *error+=error_tot[i*n+j]/pow(n,2);//(phi_guess[i*n+j]-phi_old[i*n+j])/phi_old[i*n+j]/(n*n);
                        }

#endif
		//	relative_error( phi_guess,phi_old,n,error);
		}//end of while
#ifdef PARALLEL_GPU

                hipMemcpy( phi_guess, d_phi, n*n*sizeof(double), hipMemcpyDeviceToHost );
                //relative_error( phi_guess,phi_old,n,error);
                hipFree(d_phi);
                hipFree(d_error);
                hipFree(d_rho);
                hipFree(d_result);
#endif

	}
#ifdef DEBUG
	tr = omp_get_wtime()-tr;
	
#endif


//	print(error_tot,n);
	if( *conv_criterion>1.0 ) {
		printf( "[N = %4d                ] Finish relaxation. Total iteration = %g, final conv error = %e \n", n, *itera, *error);//*error);
	} else {
		printf("Exact solver by relaxation terminated. Total iteration = %g, final conv error = %e\n", *itera, *error);
#ifdef DEBUG
		printf("Duration of exact solver = %.3f sec. \n", tr);
#endif
	}

	free( phi_old );
	free( error );
	free( itera );
}
