#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"
#include "relative_error.h"
#include <omp.h>

extern const float L;
extern const bool sor_method; 


// arguments: (1)phi matirx, (2)rho matrix, (3)size of the matrix,(4)convergence criterion, 
// 	      (5)updating method: 1="normal", 0="even odd", (6)omega for SOR (should be 1 for smoothing => GS), 
// 	      (7)which equation are we dealing with: 0 for Poisson eq., 1 for residual eq. 

__global__
void relaxation_gpu( double (*phi_guess), double (*rho), int n, double omega, bool w, double h){
	const int i = blockIdx.x+1;
	const int j = threadIdx.x+1;
//	Compute odd cells
	if( (i%2+j%2)%2==0 ){
		 phi_guess[i*n+j] += omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
				 		 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4\\
						 -rho[i*n+j]*pow(h,2)*pow(-1,w));
	}
	__syncthreads();
	if( (i%2+j%2)%2==1 ){
                 phi_guess[i*n+j] += omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
                                                 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4
                                                 -rho[i*n+j]*pow(h,2)*pow(-1,w));
        }
}


void relaxation( double *phi_guess, double *rho, int n, double *conv_criterion, float omega, bool w ) {
#ifdef DEBUG
	double tr;
	tr	= omp_get_wtime();
#endif
//	Determine the physical grid size
	double h = L/(n-1);

//	Two end criteria for relaxation
	double *itera = (double *)malloc( sizeof(double) );
	*itera = 0;
	double *error = (double *)malloc( sizeof(double) );
	*error = 1;

//	Store the primitive input to make the comparison with the up-to-date result
	double *phi_old = (double *)malloc( n*n*sizeof(double) );

//	Set the end criterion
	double *condition1;
	double *condition2;
	if( *conv_criterion<1.0 ) {
		condition1 = error;
		condition2 = conv_criterion;
	} else {
		condition1 = conv_criterion;
		condition2 = itera;
	}

//	Relaxation
	if( sor_method==1 ) {
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//		       	copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
			for( int i=1; i<(n-1); i++ )
			for( int j=1; j<(n-1); j++ ) {
				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
			    				             + phi_guess[ind(i-1, j, n)]
							             + phi_guess[ind(i, j+1, n)]
							             + phi_guess[ind(i, j-1, n)]
						        	     - phi_guess[ind(i, j, n)]*4
						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
				*error += fabs( ( phi_guess[ind(i, j, n)] - phi_old[ind(i, j, n)] ) / phi_old[ind(i, j, n)] );
			}
		}
	} else if( sor_method==0 ) {
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//	       		copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
#ifdef PARALLEL_GPU
			double (*d_phi_guess), (*d_rho);//, (*d_error);
		//	hipMalloc( &d_phi_old, n*n*sizeof(double));
			hipMalloc( &d_phi_guess, n*n*sizeof(double));
			hipMalloc( &d_rho, n*n*sizeof(double));
		//	hipMalloc( &d_error, sizeof(double));
			hipMemcpy( d_phi_guess, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );
			hipMemcpy( d_rho, rho, n*n*sizeof(double), hipMemcpyHostToDevice );
			relaxation_gpu <<< n-2,n-2 >>> ( d_phi_guess, d_rho, n, omega, w, h);
			hipMemcpy( phi_guess, d_phi_guess, n*n*sizeof(double), hipMemcpyDeviceToHost );
			hipFree(d_rho);
                	hipFree(d_phi_guess);
                //	hipFree(d_phi_old);
		//	hipFree(d_error);
		//	hipMemcpy( error, d_error, sizeof(double), hipMemcpyDeviceToHost );
		//	printf("Using GPU.\n");
#endif

#ifdef WO_OMP
		//	printf("Not Using GPU.\n");
//			update odd part
			for( int i=1; i<(n-1); i++ )
 			for( int j=( i%2 + (i+1)%2*2 ); j<(n-1); j+=2 ) {
 				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
 				    			             + phi_guess[ind(i-1, j, n)]
 							             + phi_guess[ind(i, j+1, n)]
 							             + phi_guess[ind(i, j-1, n)]
 							             - phi_guess[ind(i, j, n)]*4
 						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
 			}
//			update even part
 			for( int i=1; i<(n-1); i++ )
 			for( int j=( (i+1)%2 + i%2*2 ); j<(n-1); j+=2 ) {
 				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
 				    			             + phi_guess[ind(i-1, j, n)]
 							             + phi_guess[ind(i, j+1, n)]
 							             + phi_guess[ind(i, j-1, n)]
 							             - phi_guess[ind(i, j, n)]*4
 						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
 			}
#endif	
			relative_error( phi_guess, phi_old, n, error );
		}//end of while
	}
#ifdef DEBUG	

	tr = omp_get_wtime()-tr;
	
#endif
	if( *conv_criterion>1.0 ) {
		printf( "[N = %4d                ] Finish relaxation. Total iteration = %g, final conv error = %e \n", n, *itera, *error);
	} else {
		printf("Exact solver by relaxation terminated. Total iteration = %g, final conv error = %e\n", *itera, *error);
#ifdef DEBUG
		printf("Duration of exact solver = %.3f sec. \n", tr);
#endif
	}

	free( phi_old );
	free( error );
	free( itera );
}
