#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"
#include "relative_error.h"
#include <omp.h>

extern const float L;
extern const bool sor_method; 


// arguments: (1)phi matirx, (2)rho matrix, (3)size of the matrix,(4)convergence criterion, 
// 	      (5)updating method: 1="normal", 0="even odd", (6)omega for SOR (should be 1 for smoothing => GS), 
// 	      (7)which equation are we dealing with: 0 for Poisson eq., 1 for residual eq. 

__global__
void relaxation_gpu_odd( double (*phi_guess), double (*rho), int n, double omega, bool w, double h ){
	int job = n/BLOCK_SIZE+1;

	for( int a=0;a<job;a++ )
	for( int b=0;b<job;b++ ){
	const int i = blockIdx.x*job+a+1;
	const int j = threadIdx.x*job+b+1;
		if( i<n-1 && j<n-1 ){
//	Compute odd cells
		if( (i%2+j%2)%2==0 ){
		 double r = omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
				 		 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4\\
						 -rho[i*n+j]*pow(h,2)*pow(-1,w));
		 //error[i*n+j]     = fabs(r/(phi_guess[i*n+j]*pow(n,2)));
		 phi_guess[i*n+j] += r;
		}
		}
	}
}
__global__
void relaxation_gpu_even( double (*phi_guess), double (*rho), int n, double omega, bool w, double h ){
        int job = n/BLOCK_SIZE+1;

        for( int a=0;a<job;a++ )
        for( int b=0;b<job;b++ ){
        const int i = blockIdx.x*job+a+1;
        const int j = threadIdx.x*job+b+1;
                if( i<n-1 && j<n-1 ){
//      Compute odd cells
                if( (i%2+j%2)%2==1 ){
                 double r = omega/4 * ( phi_guess[(i+1)*n+j]+ phi_guess[(i-1)*n+j]
                                                 +phi_guess[i*n+(j+1)]+ phi_guess[i*n+(j-1)]-phi_guess[i*n+j]*4\\
                                                 -rho[i*n+j]*pow(h,2)*pow(-1,w));
                 //error[i*n+j]     = fabs(r/(phi_guess[i*n+j]*pow(n,2)));
                 phi_guess[i*n+j] += r;
                }
                }
        }
}

__global__
void compute_error( double (*error), int n, double (*result)){
	int i = threadIdx.x;
	for( int j=1;j<n;j++ )	error[i*n]+=error[i*n+j];
	__syncthreads();
	if( threadIdx.x == 0 ){
		*result = 0.0;
		for( int k=0;k<n;k++ ) *result += error[k*n];
	}
}

__global__
void relative_error_gpu( double *expe, double *theo, int n, double *error ) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	error[i*n+j] = fabs( ( expe[i*n+j] - theo[i*n+j] ) / theo[i*n+j] );
}




void relaxation( double *phi_guess, double *rho, int n, double *conv_criterion, float omega, bool w ) {
#ifdef DEBUG
	double tr;
	tr	= omp_get_wtime();
#endif
//	Determine the physical grid size
	double h = L/(n-1);

//	Two end criteria for relaxation
	double *itera = (double *)malloc( sizeof(double) );
	*itera = 0;
	double *error = (double *)malloc( sizeof(double) );
	*error = 1;

//	Store the primitive input to make the comparison with the up-to-date result
	double *phi_old = (double *)malloc( n*n*sizeof(double) );

//	Set the end criterion
	double *condition1;
	double *condition2;
	if( *conv_criterion<1.0 ) {
		condition1 = error;
		condition2 = conv_criterion;
	} else {
		condition1 = conv_criterion;
		condition2 = itera;
	}

//	Relaxation
	if( sor_method==1 ) {
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//		       	copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
			for( int i=1; i<(n-1); i++ )
			for( int j=1; j<(n-1); j++ ) {
				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
			    				             + phi_guess[ind(i-1, j, n)]
							             + phi_guess[ind(i, j+1, n)]
							             + phi_guess[ind(i, j-1, n)]
						        	     - phi_guess[ind(i, j, n)]*4
						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
				*error += fabs( ( phi_guess[ind(i, j, n)] - phi_old[ind(i, j, n)] ) / phi_old[ind(i, j, n)] );
			}
		}
	} else if( sor_method==0 ) {
#ifdef PARALLEL_GPU
		double (*d_phi), (*d_rho);//, (*d_error), (*d_result), (*d_phi_old);
#endif
		while( *condition1 > *condition2 ) {
			*itera += 1;
			*error = 0;
//	       		copy old potential
			memcpy( phi_old, phi_guess, n*n*sizeof(double) );
#ifdef PARALLEL_GPU
			double *error_tot;
			error_tot = (double *)malloc(n*n*sizeof(double));
			hipMalloc( &d_phi, n*n*sizeof(double));
			//hipMalloc( &d_phi_old, n*n*sizeof(double));
			hipMalloc( &d_rho, n*n*sizeof(double));
			//hipMalloc( &d_error, n*n*sizeof(double));
			//hipMalloc( &d_result, sizeof(double));
			//hipMemcpy( d_phi_old, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );
			hipMemcpy( d_phi, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );
			hipMemcpy( d_rho, rho, n*n*sizeof(double), hipMemcpyHostToDevice );
			relaxation_gpu_odd <<< BLOCK_SIZE,GRID_SIZE >>> ( d_phi, d_rho, n, omega, w, h);// d_error);
			relaxation_gpu_even <<< BLOCK_SIZE,GRID_SIZE >>> ( d_phi, d_rho, n, omega, w, h);// d_error);
		//	relative_error_gpu <<<n,n>>> (d_phi,d_phi_old,n,d_error);
		//	compute_error	<<<1,n>>> ( d_error, n, d_result);
		//	hipMemcpy( error, d_result, sizeof(double), hipMemcpyDeviceToHost );
			hipMemcpy( phi_guess, d_phi, n*n*sizeof(double), hipMemcpyDeviceToHost );
		//	hipMemcpy( error_tot, d_error, n*n*sizeof(double), hipMemcpyDeviceToHost );
		/*	for( int i=1;i<(n-1);i++ )
			for( int j=1;j<(n-1);j++ ){
				*error+=error_tot[i*n+j]/pow(n,2);//(phi_guess[i*n+j]-phi_old[i*n+j])/phi_old[i*n+j]/(n*n);
			}*/
			hipFree(d_phi);
	                //hipFree(d_error);
        	        hipFree(d_rho);
		//	printf("Using GPU.\n");
#endif

#ifdef WO_OMP
		//	printf("Not Using GPU.\n");
			
//			update odd part
			for( int i=1; i<(n-1); i++ )
 			for( int j=( i%2 + (i+1)%2*2 ); j<(n-1); j+=2 ) {
 				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
 				    			             + phi_guess[ind(i-1, j, n)]
 							             + phi_guess[ind(i, j+1, n)]
 							             + phi_guess[ind(i, j-1, n)]
 							             - phi_guess[ind(i, j, n)]*4
 						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
 			}
//			update even part
 			for( int i=1; i<(n-1); i++ )
 			for( int j=( (i+1)%2 + i%2*2 ); j<(n-1); j+=2 ) {
 				phi_guess[ind(i, j, n)] += omega/4 * ( phi_guess[ind(i+1, j, n)]
 				    			             + phi_guess[ind(i-1, j, n)]
 							             + phi_guess[ind(i, j+1, n)]
 							             + phi_guess[ind(i, j-1, n)]
 							             - phi_guess[ind(i, j, n)]*4
 						        	     - rho[ind(i, j, n)] * pow(h,2) * pow(-1,w) );
 			}
#endif
			relative_error( phi_guess, phi_old, n, error );
//#endif
		}//end of while
#ifdef PARALLE_GPU
//		hipMemcpy( phi_guess, d_phi_old, n*n*sizeof(double), hipMemcpyDeviceToHost );
//		hipFree(d_phi_old);
 //       	hipFree(d_error);
//		hipFree(d_rho);
#endif
	}
#ifdef DEBUG	

	tr = omp_get_wtime()-tr;
	
#endif
	if( *conv_criterion>1.0 ) {
		printf( "[N = %4d                ] Finish relaxation. Total iteration = %g, final conv error = %e \n", n, *itera, *error);
	} else {
		printf("Exact solver by relaxation terminated. Total iteration = %g, final conv error = %e\n", *itera, *error);
#ifdef DEBUG
		printf("Duration of exact solver = %.3f sec. \n", tr);
#endif
	}

	free( phi_old );
	free( error );
	free( itera );
}
