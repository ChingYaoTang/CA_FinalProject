#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include "basic.h"

extern const float L;

// input matrices and their size, calculate residual by exact L
// res = L*phi_guess "-" rho.
// For original Poisson equation, this is just the case;
// but for successive residual equation, we must impose an extra minus sign on rho.


// arguments: (1)phi_guess matrix, (2)rho matrix, (3)residual matrix, (4)matrix size, (5)which equation are we dealing with
__global__
void residual_gpu( double *phi_guess, double *rho, double *residual, int n, bool w , double h) {
	const int i = blockIdx.x+1;
        const int j = threadIdx.x+1;
	residual[i*n+j] = 1/pow(h,2) * ( phi_guess[(i+1)*n+j]
                                       + phi_guess[(i-1)*n+j]
                                       + phi_guess[i*n+(j+1)]
                                       + phi_guess[i*n+(j-1)]
                                       - phi_guess[i*n+j]*4 )
                                       - rho[i*n+j]*pow(-1,w);
}

__global__
void zero_gpu( double *residual, int n){
	const int i = blockIdx.x;
        const int j = threadIdx.x;
        residual[i*n+j] = 0.0;
}



void cal_residual( double *phi_guess, double *rho, double *residual, int n, bool w ) {
	double h = L/(n-1);
/*
#ifdef GPU
	double *d_residual,*d_phi_guess,*d_rho;
	hipMalloc( &d_residual, n*n*sizeof(double));
	hipMalloc( &d_phi_guess, n*n*sizeof(double));
	hipMalloc( &d_rho,  n*n*sizeof(double));
	hipMemcpy( d_phi_guess, phi_guess, n*n*sizeof(double), hipMemcpyHostToDevice );	
	hipMemcpy( d_rho, rho, n*n*sizeof(double), hipMemcpyHostToDevice );
	zero_gpu     <<<n,n>>> ( d_residual, n);
	residual_gpu <<<n-2,n-2>>> ( d_phi_guess, d_rho, d_residual, n, w ,h);
	hipMemcpy( residual, d_residual, n*n*sizeof(double), hipMemcpyDeviceToHost );
	hipFree(d_residual);
	hipFree(d_phi_guess);
	hipFree(d_rho);
#else*/
//      calculate interior points
//      0 for original Poisson equation, 1 for residual equation
#	ifdef OPENMP
#	pragma omp parallel for collapse(2)
#	endif
	for( int i=1; i<n-1; i++ )
	for( int j=1; j<n-1; j++ ) {
		residual[ind(i, j, n)] = 1/pow(h,2) * ( phi_guess[ind(i+1, j, n)]
	  				 	      + phi_guess[ind(i-1, j, n)]
					 	      + phi_guess[ind(i, j+1, n)]
						      + phi_guess[ind(i, j-1, n)]
						      - phi_guess[ind(i, j, n)]*4 )
				         - rho[ind(i, j, n)]*pow(-1,w);
	}

//	impose homogeneous boundary condition
#	ifdef OPENMP
#	pragma omp parallel for
#	endif
	for( int i=0; i<n; i++ ) {
		residual[ind(i, 0, n)] = residual[ind(i, n-1, n)] = residual[ind(0, i, n)] = residual[ind(n-1, i, n)] = 0.0;
	}
//#endif
#	ifdef DEBUG
	printf("[N = %4d                ] Finish residual calculation.\n", n);
#	endif

}
