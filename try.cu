
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#define N		11
#define GRID_SIZE	N
#define BLOCK_SIZE  	N


int ind( int i, int j, int NGrid ) {
        return i * NGrid + j;
}

void print( double *matrix, int n) {
    for( int i=0; i<n; i++ ) {
        for( int j=0; j<n; j++ ) {
            int index = i*n + j;
                printf("%.3f\t", matrix[index]);
        }
    printf("\n");
    }
}

__global__
void restriction_gpu( double (*matrix_f), int n_f, double (*matrix_c) ){
//      const int p = blockDim.x*blockIdx.x + threadIdx.x;
        int n_c = (n_f+1)/2;
        //int p   = blockDim.x*blockIdx.x + threadIdx.x;
        int i_c = blockIdx.x;
        int j_c = threadIdx.x;
        int i_f = 2*i_c;
        int j_f = 2*j_c;
        if( i_c<n_c && j_c<n_c){
        matrix_c[i_c*n_c+j_c] = matrix_f[i_c*n_c+j_c]/4
                              + ( matrix_f[(i_f+1)*n_f+j_f]
                                + matrix_f[(i_f-1)*n_f+j_f]
                                + matrix_f[i_f*n_f+(j_f+1)]
                                + matrix_f[i_f*n_f+(j_f-1)] )/8
                                + ( matrix_f[(i_f+1)*n_f+(j_f+1)]
                                  + matrix_f[(i_f-1)*n_f+(j_f-1)]
                                  + matrix_f[(i_f+1)*n_f+(j_f-1)]
                                  + matrix_f[(i_f-1)*n_f+(j_f+1)] )/16;
        }
}




int main( void  ){
	printf( "test restriction\n" );
        double *phi_corr_h_ = (double *)malloc( N * N * sizeof(double) );
        for( int i=0; i<N; i++) {
                for( int j=0; j<N; j++) {
                        if( i==0 || j==0 || i==N-1 || j==N-1) phi_corr_h_[ind( i, j, N )] = 0.0;
                        else phi_corr_h_[ind( i, j, N )] = 1.0;
                }
        }
        printf( "phi_corr_h\n" );
        print( phi_corr_h_, N );
        double *phi_corr_2h_ = (double *)malloc( (N+1)/2 * (N+1)/2 * sizeof(double) );
        

	int n_f = N;
	int n_c = (n_f+1)/2;
//	restriction( phi_corr_h_, N, phi_corr_2h_ );
       	double (*d_matrix_f),(*d_matrix_c);
        hipMalloc( &d_matrix_f, n_f*n_f*sizeof(double));
        hipMalloc( &d_matrix_c, n_c*n_c*sizeof(double));
        hipMemcpy( d_matrix_f, phi_corr_h_, n_f*n_f*sizeof(double), hipMemcpyHostToDevice );
        restriction_gpu  <<< GRID_SIZE, BLOCK_SIZE >>> ( d_matrix_f, n_f, d_matrix_c );
        hipMemcpy( phi_corr_2h_, d_matrix_c, n_c*n_c*sizeof(double), hipMemcpyDeviceToHost );
        hipFree(d_matrix_f);
        hipFree(d_matrix_c);
        printf("Using gpu restrict.\n");
 
	
	printf( "phi_corr_2h after restriction\n" );
        print( phi_corr_2h_, (N+1)/2 );
        free(phi_corr_h_);
        free(phi_corr_2h_);
	
	return 0;
}
