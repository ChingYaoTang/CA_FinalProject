#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include "basic.h"
#include <omp.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


// Input the fine gird matrix and calculate the corresponding coarse grid matrix by full weighting operator

// arguments: (1)fine matrix, (2)matrix size of fine matrix, (3)coarse matrix

__global__
void restriction_gpu( double (*matrix_f), int n_f, double (*matrix_c) ){
//	const int p = blockDim.x*blockIdx.x + threadIdx.x;
	int n_c = (n_f+1)/2;
	int p   = blockDim.x*blockIdx.x + threadIdx.x;
	int i_c = p/n_c;
	int j_c = p%n_c;
	int i_f = 2*i_c;
	int j_f = 2*j_c;
	if( i_c<n_c && j_c<n_c){
	matrix_c[i_c*n_c+j_c] = 20.0;/*matrix_f[i_c*n_c+j_c]/4
                              + ( matrix_f[(i_f+1)*n_f+j_f]
                                + matrix_f[(i_f-1)*n_f+j_f]
                                + matrix_f[i_f*n_f+(j_f+1)]
                                + matrix_f[i_f*n_f+(j_f-1)] )/8
                                + ( matrix_f[(i_f+1)*n_f+(j_f+1)]
                                  + matrix_f[(i_f-1)*n_f+(j_f-1)]
                                  + matrix_f[(i_f+1)*n_f+(j_f-1)]
                                  + matrix_f[(i_f-1)*n_f+(j_f+1)] )/16;*/
	}
	}


void restriction( double *matrix_f, int n_f, double *matrix_c ) {
#	ifdef DEBUG
	double tr;
	tr = omp_get_wtime();
#	endif
	int n_c = (n_f+1)/2;
	int i_c, j_c, i_f, j_f;
	
#	ifdef GPU
	double (*d_matrix_f),(*d_matrix_c);
	hipMalloc( &d_matrix_f, n_f*n_f*sizeof(double));
	hipMalloc( &d_matrix_c, n_c*n_c*sizeof(double));
	hipMemcpy( d_matrix_f, matrix_f, n_f*n_f*sizeof(double), hipMemcpyHostToDevice );
	restriction_gpu  <<< GRID_SIZE, BLOCK_SIZE >>> ( d_matrix_f, n_f, d_matrix_c );
	hipMemcpy( matrix_c, d_matrix_c, n_c*n_c*sizeof(double), hipMemcpyDeviceToHost );
	hipFree(d_matrix_f);
	hipFree(d_matrix_c);
	printf("Using gpu restrict.\n");
#	endif
	
//	Interior points
#	ifdef OPENMP
#	pragma omp parallel for collapse( 2 ) private( i_f, j_f )
//#	endif
	for( i_c=1; i_c<n_c; i_c++ )
	for( j_c=1; j_c<n_c; j_c++ ) {
		i_f = 2*i_c;
		j_f = 2*j_c;
#		ifdef	FULL_WEIGHTING
		matrix_c[ind(i_c, j_c, n_c)] = matrix_f[ind(i_f, j_f, n_f)]/4
				               + ( matrix_f[ind(i_f+1, j_f, n_f)]
				                 + matrix_f[ind(i_f-1, j_f, n_f)]
			                         + matrix_f[ind(i_f, j_f+1, n_f)]
				                 + matrix_f[ind(i_f, j_f-1, n_f)] )/8
				               + ( matrix_f[ind(i_f+1, j_f+1, n_f)]
		 		                 + matrix_f[ind(i_f-1, j_f-1, n_f)]
			 	                 + matrix_f[ind(i_f+1, j_f-1, n_f)]
				                 + matrix_f[ind(i_f-1, j_f+1, n_f)] )/16;
#		endif
#		ifdef	HALF_WEIGHTING
		matrix_c[ind(i_c, j_c, n_c)] = matrix_f[ind(i_f, j_f, n_f)]/2
				               + ( matrix_f[ind(i_f+1, j_f, n_f)]
				                 + matrix_f[ind(i_f-1, j_f, n_f)]
			                         + matrix_f[ind(i_f, j_f+1, n_f)]
				                 + matrix_f[ind(i_f, j_f-1, n_f)] )/8;
#		endif
	}
	printf("using openmp");
#endif
//	Boundary points
#	ifdef OPENMP
#	pragma omp parallel for private( i_f )
#	endif
	for( i_c=0; i_c<n_c; i_c++  ) {
		i_f = i_c*2;
//	Up & down boundaries
		matrix_c[ind(i_c, 0, n_c)]     = matrix_f[ind(i_f, 0, n_f)];
		matrix_c[ind(i_c, n_c-1, n_c)] = matrix_f[ind(i_f, n_f-1, n_f)];
		
//	Left & right boundaries
		matrix_c[ind(0, i_c, n_c)]     = matrix_f[ind(0, i_f, n_f)];
		matrix_c[ind(n_c-1, i_c, n_c)] = matrix_f[ind(n_f-1, i_f, n_f)];
	}

#	ifdef DEBUG
	tr = omp_get_wtime()-tr;

#	ifdef	FULL_WEIGHTING
	printf("[N_f = %4d -> N_c = %4d] Finish full-weighting restriction.(Duration = %.3f sec) \n", n_f, n_c, tr);
#	endif
#	ifdef	HALF_WEIGHTING
	printf("[N_f = %4d -> N_c = %4d] Finish half-weighting restriction.(Duration = %.3f sec) \n", n_f, n_c, tr);
#	endif

#	endif

}


